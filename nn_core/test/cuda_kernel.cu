#include "hip/hip_runtime.h"
#include "cuda_kernel.cuh"

#ifndef __HIPCC__
#define __HIPCC__
#endif

#include <hip/device_functions.h>
#include <>


__global__ void __func(int* arr, int len) {
	cuint index = blockIdx.x * blockDim.x + threadIdx.x;

	if (index < len) arr[index] += 10;
}


vector<int> func(vector<int> arr) {
	int* h_arr = new int[arr.size()];
	int* d_arr = NULL;

	check_cuda(hipMalloc(&d_arr, sizeof(int) * arr.size()));

	for (int i = 0; i < arr.size(); ++i) h_arr[i] = arr[i];
	check_cuda(hipMemcpy(d_arr, h_arr, sizeof(int) * arr.size(), hipMemcpyHostToDevice));

	dim3 threads(32);
	dim3 blocks((arr.size() + 32 - 1) / 32);

	__func<<<blocks, threads>>>(d_arr, arr.size());

	check_cuda(hipDeviceSynchronize());
	check_cuda(hipMemcpy(h_arr, d_arr, sizeof(int) * arr.size(), hipMemcpyDeviceToHost));

	for (int i = 0; i < arr.size(); ++i) arr[i] = h_arr[i];

	delete[] h_arr;
	check_cuda(hipFree(d_arr));

	return arr;
}