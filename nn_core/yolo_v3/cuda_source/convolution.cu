#include "hip/hip_runtime.h"
#include "convolution.cuh"
#include "cuda_misc.cuh"


#ifndef __HIPCC__
#define __HIPCC__
#endif

#include <hip/device_functions.h>
#include <>



/**********************************************/
/*											  */
/*				 kernel function			  */
/*										      */
/**********************************************/
#if 0
__global__ void __conv2d(
	const nn_type* input,
	const nn_type* kernel,
	nn_type* output,
	cuint* indice,
	cuint in_h,
	cuint in_w,
	cuint k_n,
	cuint k_c,
	cuint k_h,
	cuint k_w,
	cuint out_h,
	cuint out_w,
	cuint st_h,
	cuint st_w
) {
	cuint cx = blockIdx.x * blockDim.x + threadIdx.x;
	cuint cy = blockIdx.y * blockDim.y + threadIdx.y;
	cuint sidx = threadIdx.y * BLOCK_32 + threadIdx.x;

	cuint x0 = (cx % out_w) * st_w;
	cuint y0 = (cx / out_w) * st_h;

	cuint n = k_w * k_h * k_c;
	cuint k = out_w * out_h;

	__shared__ nn_type share_in[BLOCK_32 * BLOCK_32];
	__shared__ nn_type share_k[BLOCK_32 * BLOCK_32];

	const nn_type* p_input = input + (y0 * in_w + x0);
	const nn_type* p_kernel = kernel + (cy * k_w * k_h * k_c);

	nn_type sum = 0.f;

	for (uint i = 0; i < n; i += BLOCK_32) {
		uint th_x = i + threadIdx.x;
		uint th_y = i + threadIdx.y;

		__syncthreads();

		share_k[sidx] = th_x < n && cy < k_n ? p_kernel[th_x] : 0.f;
		share_in[sidx] = cx < k && th_y < n ? p_input[indice[th_y]] : 0.f;

		__syncthreads();

#pragma unroll
		for (uint e = 0; e < BLOCK_32; ++e) {
			sum += share_in[e * BLOCK_32 + threadIdx.x] * share_k[threadIdx.y * BLOCK_32 + e];
		}
	}

	if (cx < k && cy < k_n) {
		output[cy * k + cx] = sum;
	}
}

#else

__global__ void __conv2d(
	const nn_type* input,
	const nn_type* kernel,
	nn_type* output,
	cuint* indice,
	cuint in_h,
	cuint in_w,
	cuint k_h,
	cuint k_w,
	cuint k_ic,
	cuint k_oc,
	cuint out_h,
	cuint out_w,
	cuint st_h,
	cuint st_w
) {
	cuint cx = blockIdx.x * blockDim.x + threadIdx.x;
	cuint cy = blockIdx.y * blockDim.y + threadIdx.y;												
	cuint tidx = threadIdx.y * BLOCK_32 + threadIdx.x;

	cuint out_x = cx % out_w;
	cuint out_y = cx / out_w;
	cuint in_x = out_x * st_w;
	cuint in_y = out_y * st_h;

	cuint k = k_h * k_w * k_ic;
	cuint n = out_w * out_h;

	__shared__ nn_type share_in[BLOCK_32 * BLOCK_32];
	__shared__ nn_type share_k[BLOCK_32 * BLOCK_32];

	const nn_type* p_input = input + ((in_y * in_w * k_ic) + (in_x * k_ic));
	const nn_type* p_kernel = kernel + cy;
	nn_type* p_output = output + ((out_y * out_w * k_oc) + (out_x * k_oc) + cy);

	nn_type sum = 0.f;

	for (uint i = 0; i < k; i += BLOCK_32) {
		uint th_x = i + threadIdx.x;
		uint th_y = i + threadIdx.y;

		__syncthreads();

		//share_k[tidx] = th_y < k && cy < k_oc ? p_kernel[th_y * k_oc] : 0.f;
		//share_in[tidx] = th_x < k && cx < n ? p_input[indice[th_x]] : 0.f;
		share_k[tidx] = th_x < k && cy < k_oc ? p_kernel[th_x * k_oc] : 0.f;
		share_in[tidx] = th_y < k && cx < n ? p_input[indice[th_y]] : 0.f;

		__syncthreads();

#pragma unroll
		for (uint e = 0; e < BLOCK_32; ++e) {
			//sum += share_in[threadIdx.y * BLOCK_32 + e] * share_k[e * BLOCK_32 + threadIdx.x];
			sum += share_in[e * BLOCK_32 + threadIdx.x] * share_k[threadIdx.y * BLOCK_32 + e];
		}
	}

	if (cx < n && cy < k_oc) *p_output = sum;
}

#endif


/**********************************************/
/*                                            */
/*                 NN_Conv2D                  */
/*                                            */
/**********************************************/

cuint* NN_Conv2D::set_indice(const NN_Tensor4dShape& in, const NN_Filter4dShape& k) {
	uint* h_idx = new uint[k._h * k._w * in._c];

	for (int h = 0; h < k._h; ++h) {
		cuint kh = k._w * in._c * h;
		cuint in_h = in._w * in._c * h;
		for (int w = 0; w < k._w; ++w) {
			cuint kw = in._c * w;
			cuint in_w = in._c * w;
			for (int c = 0; c < in._c; ++c) {
				h_idx[kh + kw + c] = (uint)(in_h + in_w + c);
			}
		}
	}

	cuint* g_idx = set_const_mem(h_idx, k._h * k._w * in._c, 0);

	delete[] h_idx;

	return g_idx;
}

NN_Conv2D::NN_Conv2D(cuint amounts, const NN_Shape& filter_size, const NN_Shape& stride, const std::string& pad, bool use_bias, const std::string& name) :
	_amounts(amounts),
	_filter_size(filter_size),
	_stride(stride),
	_pad(pad),
	_use_bias(use_bias),
	NN_Layer(name, "conv2d")
{
}

void NN_Conv2D::get_output_shape(const NN_List<NN_Shape>& input_shape, NN_List<NN_Shape>& output_shape) {
	const NN_Shape& shape = input_shape[0].val();

	if (_pad == "same") {
		int n = shape[0];
		int h = (int)ceil((float)shape[1] / _stride[1]);
		int w = (int)ceil((float)shape[2] / _stride[0]);
		int c = _amounts;

		output_shape.append(NN_Shape({ n, h, w, c }));
	}
	else {
		int n = shape[0];
		int h = (int)floorf((float)(shape[1] - _filter_size[1]) / _stride[1] + 1);
		int w = (int)floorf((float)(shape[2] - _filter_size[0]) / _stride[0] + 1);
		int c = _amounts;

		output_shape.append(NN_Shape({ n, h, w, c }));
	}
}

void NN_Conv2D::build(const NN_List<NN_Shape>& input_shape, NN_List<GpuTensor<nn_type>>& weights) {
	const NN_Shape& shape = input_shape[0].val();
	
	_filter.resize(NN_Shape({ _filter_size[1], _filter_size[0], shape[3], _amounts }));
	set_random_uniform(_filter, -0.1f, 0.1f);
	weights.append(_filter);

	if (_use_bias) {
		_bias = GpuTensor<nn_type>::zeros({ _amounts });
		weights.append(_bias);
	}
}

void NN_Conv2D::run(NN_Stream& st, const NN_List<GpuTensor<nn_type>>& input, NN_List<GpuTensor<nn_type>>& output) {
	const GpuTensor<nn_type>& m_input = input[0].val();
	GpuTensor<nn_type>& m_output = output[0].val();

	//std::cout << "input: " << m_input.get_shape();
	//std::cout << "kernel: " << _filter.get_shape();
	//std::cout << "output: " << m_output.get_shape();

	const NN_Tensor4dShape in = m_input.get_shape().get_4d_shape();
	const NN_Tensor4dShape out = m_output.get_shape().get_4d_shape();
	const NN_Filter4dShape k = _filter.get_shape().get_filter_shape();

	//printf("input: [%d, %d, %d, %d]\n", in.n, in.c, in.h, in.w);
	//printf("kernel: [%d, %d, %d, %d]\n", k.n, k.c, k.h, k.w);
	//printf("output: [%d, %d, %d, %d]\n", out.n, out.c, out.h, out.w);

	dim3 threads(BLOCK_32, BLOCK_32);
	dim3 blocks = get_grid_size(threads, out._w * out._h, out._c);

	const nn_type* input_data = m_input.get_ptr();
	nn_type* output_data = m_output.get_ptr();
	const nn_type* filter_data = _filter.get_ptr();

	hipStream_t* p_st = st.get_stream();

	if (_pad == "same") {
		NN_Tensor4dShape pad = in;

		if (_stride[0] == 1) pad._w = in._w - 1 + k._w;
		else pad._w = (in._w / _stride[0]) + k._w;

		if (_stride[1] == 1) pad._h = in._h - 1 + k._h;
		else pad._h = (in._h / _stride[1]) + k._h;

		cuint* c_indice = set_indice(pad, k);

		for (uint n = 0; n < (uint)in._n; ++n) {
			const nn_type* in_data = input_data + (n * in._h * in._w * in._c);
			nn_type* out_data = output_data + (n * out._h * out._w * out._c);

			if (pad._h != in._h || pad._w != in._w) {
				nn_type* pad_space = NULL;
				cuint pad_size = pad._h * pad._w * pad._c;

				check_cuda(hipMallocAsync((void**)&pad_space, sizeof(nn_type) * pad_size, p_st[n % STREAMS]));
				check_cuda(hipMemsetAsync(pad_space, 0, sizeof(nn_type) * pad_size, p_st[n % STREAMS]));

				padding_dilation(
					p_st[n % STREAMS],
					in_data,
					pad_space,
					in,
					pad,
					_stride[1] == 1 ? (pad._w - in._w) / 2 : 0,
					_stride[0] == 1 ? (pad._h - in._h) / 2 : 0,
					_stride[1],
					_stride[0]
				);
#if _DEBUG
				check_cuda(hipStreamSynchronize(st[n % STREAMS]));
				check_cuda(hipGetLastError());
#endif
				__conv2d<<<blocks, threads, 0, p_st[n % STREAMS]>>>(
					pad_space,
					filter_data,
					out_data,
					c_indice,
					(uint)pad._h,
					(uint)pad._w,
					(uint)k._h,
					(uint)k._w,
					(uint)k._in_c,
					(uint)k._out_c,
					(uint)out._h,
					(uint)out._w,
					(uint)_stride[1],
					(uint)_stride[0]
					);
#if _DEBUG
				check_cuda(hipStreamSynchronize(st[n % STREAMS]));
				check_cuda(hipGetLastError());
#endif
				hipFreeAsync(pad_space, p_st[n % STREAMS]);
			}
			else {
				__conv2d<<<blocks, threads, 0, p_st[n % STREAMS]>>>(
					in_data,
					filter_data,
					out_data,
					c_indice,
					(uint)in._h,
					(uint)in._w,
					(uint)k._h,
					(uint)k._w,
					(uint)k._in_c,
					(uint)k._out_c,
					(uint)out._h,
					(uint)out._w,
					(uint)_stride[1],
					(uint)_stride[0]
				);
#if _DEBUG
				check_cuda(hipStreamSynchronize(st[n % STREAMS]));
				check_cuda(hipGetLastError());
#endif
			}
		}
	}
	else {
		cuint* c_indice = set_indice(in, k);

		for (uint n = 0; n < (uint)in._n; ++n) {
			const nn_type* in_data = input_data + (n * in._c * in._h * in._w);
			nn_type* out_data = output_data + (n * out._c * out._h * out._w);

			__conv2d<<<blocks, threads, 0, p_st[n % STREAMS]>>>(
				in_data,
				filter_data,
				out_data,
				c_indice,
				(uint)in._h,
				(uint)in._w,
				(uint)k._h,
				(uint)k._w,
				(uint)k._in_c,
				(uint)k._out_c,
				(uint)out._h,
				(uint)out._w,
				(uint)_stride[1],
				(uint)_stride[0]
				);
#if _DEBUG
			check_cuda(hipStreamSynchronize(p_st[n % STREAMS]));
			check_cuda(hipGetLastError());
#endif
		}
		//Tensor<nn_type> tmp(_filter.get_shape());
		//tmp = _filter;

		//std::cout << std::endl << tmp;
	}
	if (_use_bias) {
		add_bias_2d(st, m_output, _bias, m_output);
#if _DEBUG
		check_cuda(hipDeviceSynchronize());
		check_cuda(hipGetLastError());
#endif
	}
}

NN_Backward* NN_Conv2D::create_backward(std::vector<bool>& mask) {
	return new NN_dConv2D(*this);
}

NN_List<GpuTensor<nn_type>> NN_Conv2D::get_weight() {
	if (_use_bias) return { _filter, _bias };
	else return { _filter, };
}


/**********************************************/
/*                                            */
/*                 NN_dConv2D                 */
/*                                            */
/**********************************************/

NN_dConv2D::NN_dConv2D(NN_Conv2D& layer) :
	NN_Backward_t(layer)
{
}

void NN_dConv2D::run(
	NN_Stream& st,
	const NN_List<GpuTensor<nn_type>>& input,
	const NN_List<GpuTensor<nn_type>>& doutput,
	NN_List<GpuTensor<nn_type>>& dinput
) {

}

NN_Optimizer* NN_dConv2D::create_optimizer(const NN_Optimizer& optimizer) {
	return optimizer.create({ _layer._filter, _layer._bias });
}


/**********************************************

			     KernelConv2d

**********************************************/
/*
void kernel_conv2d(
	const nn_type* d_output,
	const nn_type* input,
	nn_type* grad,
	const nn_shape& out_shape,
	const nn_shape& in_shape,
	const nn_shape& grad_shape
) {
	cint hw = out_shape[2] * out_shape[3];
	uint* indice = NULL;
	uint* _indice = new uint[hw];

	for (int h = 0; h < out_shape[2]; ++h) {
		for (int w = 0; w < out_shape[3]; ++w) {
			_indice[h * out_shape[3] + w] = h * in_shape[3] + w;
		}
	}

	if (hw > CONST_ELEM_SIZE) {
		check_cuda(hipMalloc(&indice, sizeof(uint) * hw));
		check_cuda(hipMemcpy(indice, _indice, sizeof(uint) * hw, hipMemcpyHostToDevice));
	}
	else {
		indice = get_indice_ptr();
		set_indice(_indice, sizeof(uint) * hw, 0);
	}
	delete[] _indice;

	dim3 threads(BLOCK_32, BLOCK_32);
	dim3 blocks = get_grid_size(threads, grad_shape[1] * grad_shape[2] * grad_shape[3], grad_shape[0]);

	for (int i = 0; i < out_shape[0]; ++i) {
		const nn_type* d_dout = d_output + (i * out_shape[1] * out_shape[2] * out_shape[3]);
		const nn_type* d_input = input + (i * in_shape[1] * in_shape[2] * in_shape[3]);

		__kernel_conv2d<<<blocks, threads>>>(
			indice,
			d_dout,
			d_input,
			grad,
			out_shape[1],
			out_shape[2],
			out_shape[3],
			in_shape[1],
			in_shape[2],
			in_shape[3],
			grad_shape[2],
			grad_shape[3]
		);
	}

	if (hw > CONST_ELEM_SIZE) check_cuda(hipFree(indice));
}
*/