#include "hip/hip_runtime.h"
#define CUDA_API_PER_THREAD_DEFAULT_STEAM 
#include "matmul.cuh"
#include "cuda_misc.cuh"


#ifndef __HIPCC__
#define __HIPCC__
#endif

#include <hip/device_functions.h>
#include <>


/**********************************************/
/*											  */
/*				 kernel function			  */
/*										      */
/**********************************************/

__global__ void __matmul(
	const nn_type* a,
	const nn_type* b,
	nn_type* c,
	cuint m,
	cuint k,
	cuint n
) {
	cuint cx = blockIdx.x * blockDim.x + threadIdx.x;
	cuint cy = blockIdx.y * blockDim.y + threadIdx.y;

	__shared__ nn_type sm_a[BLOCK_32 * BLOCK_32];
	__shared__ nn_type sm_b[BLOCK_32 * BLOCK_32];

	cuint tidx = threadIdx.y * BLOCK_32 + threadIdx.x;
	nn_type val = 0.f;

	for (uint i = 0; i < k; i += BLOCK_32) {
		__syncthreads();

		sm_a[tidx] = (threadIdx.x + i) < k && cy < m ? a[cy * k + (threadIdx.x + i)] : 0.f;
		sm_b[tidx] = cx < n && (threadIdx.y + i) < k ? b[(threadIdx.y + i) * n + cx] : 0.f;

		__syncthreads();

#pragma unroll
		for (uint e = 0; e < BLOCK_32; ++e) {
			val += sm_a[threadIdx.y * BLOCK_32 + e] * sm_b[e * BLOCK_32 + threadIdx.x];
		}
	}

	if (cx < n && cy < m) {
		c[cy * n + cx] = val;
	}
}


/**********************************************/
/*                                            */
/*                   NN_Dense                 */
/*                                            */
/**********************************************/

NN_Dense::NN_Dense(const int amounts, const char* name) :
	NN_Layer(name),
	_amounts(amounts)
{
}

void NN_Dense::get_output_shape(const std::vector<NN_Shape>& input_shape, std::vector<NN_Shape>& output_shape) {
	const NN_Shape& shape = input_shape[0];

	output_shape.push_back({ shape[0], _amounts });
}

void NN_Dense::build(const std::vector<NN_Shape>& input_shape) {
	const NN_Shape& shape = input_shape[0];


	_weight = GpuTensor<nn_type>({ shape[1], _amounts });
	_bias = GpuTensor<nn_type>(NN_Shape({ _amounts }));
	set_random_uniform(_weight, -0.1f, 0.1f);

	Tensor<nn_type> tmp(NN_Shape({ _amounts }));
	tmp = 0.f;
	_bias = tmp;
}

void NN_Dense::run_forward(NN_Stream& st, const std::vector<GpuTensor<nn_type>>& input, std::vector<GpuTensor<nn_type>>& output) {
	const GpuTensor<nn_type>& m_input = input[0];
	GpuTensor<nn_type>& m_output = output[0];

	const NC in = m_input.get_shape().get_nc();
	const NC out = m_output.get_shape().get_nc();

	dim3 threads(BLOCK_32, BLOCK_32);
	dim3 blocks = get_grid_size(threads, out.c, out.n);

	__matmul<<<blocks, threads>>>(
		m_input.get_ptr(),
		_weight.get_ptr(),
		m_output.get_ptr(),
		in.n,
		in.c,
		out.c
	);

	add_bias_1d(m_output, _bias, m_output);
}