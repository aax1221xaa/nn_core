#include "hip/hip_runtime.h"
#include "relu.cuh"

#ifndef __HIPCC__
#define __HIPCC__
#endif

//#include <hip/device_functions.h>
#include <>


/**********************************************/
/*											  */
/*				 kernel function			  */
/*										      */
/**********************************************/

__global__ void __relu(
	const float* a,
	float* b,
	cuint length
) {
	uint cx = blockIdx.x * blockDim.x + threadIdx.x;
	
	if (cx < length) {
		b[cx] = __max(0.f, a[cx]);
	}
}

__global__ void __d_relu(
	const float* a,
	const float* b,
	float* c,
	cuint len
) {
	cuint cx = blockIdx.x * blockDim.x + threadIdx.x;

	if (cx < len && b[cx] > 0) c[cx] = a[cx];
}


/**********************************************

					  ReLU

**********************************************/

void relu(
	const nn_type* input,
	nn_type* output,
	const nn_shape& in_shape
) {
	cuint len = in_shape[0] * in_shape[1] * in_shape[2] * in_shape[3];
	dim3 threads(BLOCK_1024);
	dim3 blocks = get_grid_size(threads, len);

	__relu<<<blocks, threads>>>(
		input,
		output,
		len
	);
}

/**********************************************

					 D_ReLU

**********************************************/

void d_relu(
	const nn_type* d_output,
	const nn_type* input,
	nn_type* d_input,
	const nn_shape& in_shape
) {
	cuint len = in_shape[0] * in_shape[1] * in_shape[2] * in_shape[3];
	dim3 threads(BLOCK_1024);
	dim3 blocks = get_grid_size(threads, len);

	__d_relu<<<blocks, threads>>>(
		d_output,
		input,
		d_input,
		len
	);
}