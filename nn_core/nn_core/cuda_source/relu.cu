#include "hip/hip_runtime.h"
#include "relu.cuh"

#ifndef __HIPCC__
#define __HIPCC__
#endif

#include <hip/device_functions.h>
#include <>


/**********************************************/
/*											  */
/*				 kernel function			  */
/*										      */
/**********************************************/

__global__ void __relu(
	float* a,
	float* b,
	const uint length
) {
	uint cx = blockIdx.x * blockDim.x + threadIdx.x;
	
	if (cx < length) {
		b[cx] = __max(0.f, a[cx]);
	}
}



/**********************************************/
/*											  */
/*				  host function 			  */
/*										      */
/**********************************************/

void relu(
	hipStream_t stream,
	const CudaTensor input,
	CudaTensor output
) {
	uint input_size = get_elem_size(input);
	uint output_size = get_elem_size(output);

	if (input_size != output_size) {
		ErrorExcept("[relu] invalid input and output size. %d != %d", input_size, output_size);
	}

	uint length = input.h * input.w * input.c;
	dim3 threads(SQR_BLOCK_SIZE);
	dim3 blocks(get_grid_size(threads, length));

	for (int i = 0; i < input.n; ++i) {
		float* d_in = input.data + (i * length);
		float* d_out = output.data + (i * length);

		__relu<<<blocks, threads, 0, stream>>>(
			d_in,
			d_out,
			length
		);
	}
	check_cuda(hipStreamSynchronize(stream));
}