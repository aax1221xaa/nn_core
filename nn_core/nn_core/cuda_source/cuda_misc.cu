#include "hip/hip_runtime.h"
#include "cuda_misc.cuh"

#ifndef __HIPCC__
#define __HIPCC__
#endif

#include <hip/device_functions.h>
#include <>


/*******************************************
											  
			   kernel functions			  

*******************************************/

__global__ void __transpose(
	const float* input,
	float* output,
	cuint n,
	cuint c,
	cuint h,
	cuint w
) {
	cuint tidx = blockIdx.x * blockDim.x + threadIdx.x;
	cuint k_idx = tidx % (w * h);
	cuint k_count = tidx / (w * h);

	cuint row = k_count % c;
	cuint col = k_count / c;

	float* p_out = output + (row * (w * h * n) + col * (w * h));

	if (tidx < (n * h * w * c)) {
		p_out[k_idx] = input[tidx];
	}
}

__global__ void __padding_dilation_2d(
	const float* input,
	float* output,
	cuint in_w,
	cuint in_h,
	cuint in_c,
	cuint out_w,
	cuint out_h,
	cuint stride_x,
	cuint stride_y,
	cuint offset_x,
	cuint offset_y
) {
	cuint in_cx = blockIdx.x * blockDim.x + threadIdx.x;
	cuint in_cy = blockIdx.y * blockDim.y + threadIdx.y;
	cuint in_cz = blockIdx.z;

	cuint out_cx = in_cx * stride_x + offset_x;
	cuint out_cy = in_cy * stride_y + offset_y;

	if (in_cx < in_w && in_cy < in_h) {
		output[in_cz * (out_w * out_h) + out_cy * out_w + out_cx] = input[in_cz * (in_w * in_h) + in_cy * in_w + in_cx];
	}
}

__global__ void __add_bias_32x32(
	const float* data_a,
	const float* data_b,
	float* data_c,
	cuint n,
	cuint c
) {
	cuint cx = blockIdx.x * blockDim.x + threadIdx.x;
	cuint cy = blockIdx.y * blockDim.y + threadIdx.y;

	cuint addr = cy * c + cx;

	__shared__ float share_b[BLOCK_32];

	if (threadIdx.y == 0) share_b[threadIdx.x] = cx < c ? data_b[cx] : 0.f;
	__syncthreads();

	if (cx < c && cy < n) {
		data_c[addr] = data_a[addr] + share_b[threadIdx.x];
	}
}

__global__ void __add_bias_16x16x4(
	const float* data_a,
	const float* data_b,
	float* data_c,
	cuint c,
	cuint h,
	cuint w
) {
	cuint cx = blockIdx.x * blockDim.x + threadIdx.x;
	cuint cy = blockIdx.y * blockDim.y + threadIdx.y;
	cuint cz = blockIdx.z * blockDim.z + threadIdx.z;

	cuint addr = cz * (h * w) + cy * w + cx;

	__shared__ float share_b[BLOCK_4];

	if (threadIdx.x == 0 && threadIdx.y == 0) share_b[threadIdx.z] = cz < c ? data_b[cz] : 0.f;
	__syncthreads();

	if (cx < w && cy < h && cz < c) {
		data_c[addr] = data_a[addr] + share_b[threadIdx.z];
	}
}

__global__ void __add_bias_8x8x16(
	const float* data_a,
	const float* data_b,
	float* data_c,
	cuint c,
	cuint h,
	cuint w
) {
	cuint cx = blockIdx.x * blockDim.x + threadIdx.x;
	cuint cy = blockIdx.y * blockDim.y + threadIdx.y;
	cuint cz = blockIdx.z * blockDim.z + threadIdx.z;

	cuint addr = cz * (h * w) + cy * w + cx;

	__shared__ float share_b[BLOCK_16];

	if (threadIdx.x == 0 && threadIdx.y == 0) share_b[threadIdx.z] = cz < c ? data_b[cz] : 0.f;
	__syncthreads();

	if (cx < w && cy < h && cz < c) {
		data_c[addr] = data_a[addr] + share_b[threadIdx.z];
	}
}

__global__ void __sum_gradient_1d(
	const float* a,
	float* b,
	cuint n,
	cuint c
) {
	cuint cx = blockIdx.x * blockDim.x + threadIdx.x;
	cuint tidx = threadIdx.y * BLOCK_32 + threadIdx.x;

	__shared__ float sm[BLOCK_32 * BLOCK_32];
	
	sm[tidx] = 0.f;
	__syncthreads();

	for (uint i = 0; i < n; i += BLOCK_32) {
		cuint cy = n + threadIdx.y;

		if (cx < c && cy < n) {
			sm[tidx] += a[cy * c + cx];
		}
	}

#pragma unroll
	for (uint i = BLOCK_32 / 2; i > 0; i /= 2) {
		cuint half_side = (threadIdx.y + i) * BLOCK_32 + threadIdx.x;

		__syncthreads();
		if (threadIdx.y < i) sm[tidx] += sm[half_side];
	}

	if (cx < c && threadIdx.y == 0) b[cx] = sm[threadIdx.x];
}

__global__ void __sum_gradient_2d(
	const float* a,
	float* b,
	cuint n,
	cuint c,
	cuint h,
	cuint w
) {
	/*
	threads = [4, 16, 16]
	blocks = [c]
	*/

	__shared__ float sm[BLOCK_4 * BLOCK_16 * BLOCK_16];

	cuint tidx = threadIdx.z * BLOCK_16 * BLOCK_16 + threadIdx.y * BLOCK_16 + threadIdx.x;
	cuint cidx = blockIdx.x * (w * h);

	sm[tidx] = 0.f;
	__syncthreads();

	for (uint z = 0; z < n; z += BLOCK_4) {
		cuint tz = z + threadIdx.z;
		cuint nidx = tz * (c * h * w);

		for (uint y = 0; y < h; y += BLOCK_16) {
			cuint ty = y + threadIdx.y;
			cuint yidx = ty * w;

			for (uint x = 0; x < w; x += BLOCK_16) {
				cuint tx = x + threadIdx.x;

				if (tz < n && ty < h && tx < w) sm[tidx] += a[nidx + cidx + yidx + tx];
			}
		}
	}

#pragma unroll
	for (uint i = BLOCK_1024 / 2; i > 0; i /= 2) {
		__syncthreads();

		if (tidx < i) sm[tidx] += sm[tidx + i];
	}

	b[blockIdx.x] = sm[0];
}

/*******************************************

			    host functions

*******************************************/

void transpose(
	const nn_type* input,
	nn_type* output,
	const nn_shape& in_shape
) {
	dim3 threads(BLOCK_1024);
	dim3 blocks = get_grid_size(threads, in_shape[0] * in_shape[1] * in_shape[2] * in_shape[3]);

	__transpose<<<blocks, threads>>>(
		input,
		output,
		in_shape[0],
		in_shape[1],
		in_shape[2],
		in_shape[3]
	);
}

void padding_dilation(
	hipStream_t s,
	const nn_type* input,
	nn_type* output,
	const nn_shape& in_shape,
	const nn_shape& out_shape,
	int offset_x,
	int offset_y,
	int stride_x,
	int stride_y
) {
	dim3 threads(BLOCK_32, BLOCK_32);
	dim3 blocks = get_grid_size(threads, in_shape[1], in_shape[2], in_shape[3]);

	__padding_dilation_2d<<<blocks, threads, 0, s>>>(
		input,
		output,
		in_shape[3],
		in_shape[2],
		in_shape[1],
		out_shape[3],
		out_shape[2],
		stride_x,
		stride_y,
		offset_x,
		offset_y
	);
}

void add_bias_1d(
	const nn_type* input,
	const nn_type* bias,
	nn_type* output,
	const nn_shape& in_shape
) {
	dim3 threads(BLOCK_32, BLOCK_32);
	dim3 blocks = get_grid_size(threads, in_shape[1], in_shape[0]);

	__add_bias_32x32<<<blocks, threads>>>(
		input,
		bias,
		output,
		in_shape[0],
		in_shape[1]
	);
}

void add_bias_2d(
	hipStream_t* s,
	const nn_type* input,
	const nn_type* bias,
	nn_type* output,
	const nn_shape& in_shape,
	const nn_shape& b_shape,
	const nn_shape& out_shape
) {

	if (in_shape[2] >= BLOCK_16 && in_shape[3] >= BLOCK_16 || in_shape[1] <= BLOCK_4) {
		dim3 threads(BLOCK_16, BLOCK_16, BLOCK_4);
		dim3 blocks = get_grid_size(threads, in_shape[3], in_shape[2], in_shape[1]);

		for (uint i = 0; i < in_shape[0]; ++i) {
			const nn_type* d_in = input + (i * in_shape[1] * in_shape[2] * in_shape[3]);
			nn_type* d_out = output + (i * in_shape[1] * in_shape[2] * in_shape[3]);

			__add_bias_16x16x4<<<blocks, threads, 0, s[i % STREAMS]>>>(
				d_in,
				bias,
				d_out,
				in_shape[1],
				in_shape[2],
				in_shape[3]
			);
		}
	}
	else {
		dim3 threads(BLOCK_8, BLOCK_8, BLOCK_16);
		dim3 blocks = get_grid_size(threads, in_shape[3], in_shape[2], in_shape[1]);

		for (uint i = 0; i < in_shape[0]; ++i) {
			const nn_type* d_in = input + (i * in_shape[1] * in_shape[2] * in_shape[3]);
			nn_type* d_out = output + (i * in_shape[1] * in_shape[2] * in_shape[3]);

			__add_bias_8x8x16<<<blocks, threads, 0, s[i % STREAMS]>>>(
				d_in,
				bias,
				d_out,
				in_shape[1],
				in_shape[2],
				in_shape[3]
			);
		}
	}
}

void sum_gradient_1d(
	const nn_type* input,
	nn_type* output,
	const nn_shape& in_shape
) {
	dim3 threads(BLOCK_32, BLOCK_32);
	dim3 blocks = get_grid_size(threads, in_shape[1]);

	__sum_gradient_1d<<<blocks, threads>>>(
		input,
		output,
		in_shape[0],
		in_shape[1]
	);
}

void sum_gradient_2d(
	const nn_type* input,
	nn_type* output,
	const nn_shape& in_shape
) {
	dim3 threads(BLOCK_16, BLOCK_16, BLOCK_4);
	dim3 blocks(in_shape[0]);

	__sum_gradient_2d<<<blocks, threads>>>(
		input,
		output,
		in_shape[0],
		in_shape[1],
		in_shape[2],
		in_shape[3]
	);
}