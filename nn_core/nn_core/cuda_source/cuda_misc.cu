#include "hip/hip_runtime.h"
#define CUDA_API_PER_THREAD_DEFAULT_STEAM 
#include "cuda_misc.cuh"

#ifndef __HIPCC__
#define __HIPCC__
#endif

#include <hip/device_functions.h>
#include <>


__constant__ uint __cmem[CONST_ELEM_SIZE];


/*******************************************
											  
			   kernel functions			  

*******************************************/

__global__ void __transpose(
	const nn_type* input,
	nn_type* output,
	cuint* c_trans_ranks,
	cuint* c_dims,
	cuint* c_steps,
	uint n_ranks,
	cuint total_size
) {
	cuint tidx = blockIdx.x * blockDim.x + threadIdx.x;
	
	uint quot = tidx;
	uint src_index = 0;

	while (n_ranks) {
		--n_ranks;

		cuint rank = c_trans_ranks[n_ranks];
		cuint dim = c_dims[rank];
		cuint curr_dim = quot % dim;

		src_index += c_steps[rank] * curr_dim;

		quot /= dim;
	}

	if (tidx < total_size) {
		output[tidx] = input[src_index];
	}
}

__global__ void __padding_dilation_2d(
	const nn_type* input,
	nn_type* output,
	cuint in_w,
	cuint in_h,
	cuint in_c,
	cuint out_w,
	cuint out_h,
	cuint stride_x,
	cuint stride_y,
	cuint offset_x,
	cuint offset_y
) {
	cuint in_cx = blockIdx.x * blockDim.x + threadIdx.x;
	cuint in_cy = blockIdx.y * blockDim.y + threadIdx.y;
	cuint in_cz = blockIdx.z;

	cuint out_cx = in_cx * stride_x + offset_x;
	cuint out_cy = in_cy * stride_y + offset_y;

	if (in_cx < in_w && in_cy < in_h) {
		output[(out_cy * out_w * in_c) + (out_cx * in_c) + in_cz] = input[(in_cy * in_w * in_c) + (in_cx * in_c) + in_cz];
	}
}

__global__ void __add_bias_32x32(
	const nn_type* data_a,
	const nn_type* data_b,
	nn_type* data_c,
	cuint n,
	cuint c
) {
	cuint cx = blockIdx.x * blockDim.x + threadIdx.x;
	cuint cy = blockIdx.y * blockDim.y + threadIdx.y;

	cuint addr = cy * c + cx;

	__shared__ nn_type share_b[BLOCK_32];

	if (threadIdx.y == 0) share_b[threadIdx.x] = cx < c ? data_b[cx] : 0.f;
	__syncthreads();

	if (cx < c && cy < n) {
		data_c[addr] = data_a[addr] + share_b[threadIdx.x];
	}
}

__global__ void __add_bias_16x16x4(
	const nn_type* data_a,
	const nn_type* data_b,
	nn_type* data_c,
	cuint h,
	cuint w,
	cuint c
) {
	cuint cx = blockIdx.x * blockDim.x + threadIdx.x;
	cuint cy = blockIdx.y * blockDim.y + threadIdx.y;
	cuint cz = blockIdx.z * blockDim.z + threadIdx.z;

	cuint addr = (cy * w * c) + (cx * c) + cz;

	__shared__ nn_type share_b[BLOCK_4];

	if (threadIdx.x == 0 && threadIdx.y == 0) share_b[threadIdx.z] = cz < c ? data_b[cz] : 0.f;
	__syncthreads();

	if (cx < w && cy < h && cz < c) {
		data_c[addr] = data_a[addr] + share_b[threadIdx.z];
	}
}

__global__ void __add_bias_8x8x16(
	const nn_type* data_a,
	const nn_type* data_b,
	nn_type* data_c,
	cuint h,
	cuint w,
	cuint c
) {
	cuint cx = blockIdx.x * blockDim.x + threadIdx.x;
	cuint cy = blockIdx.y * blockDim.y + threadIdx.y;
	cuint cz = blockIdx.z * blockDim.z + threadIdx.z;

	cuint addr = (cy * w * c) + (cx * c) + cz;

	__shared__ nn_type share_b[BLOCK_16];

	if (threadIdx.x == 0 && threadIdx.y == 0) share_b[threadIdx.z] = cz < c ? data_b[cz] : 0.f;
	__syncthreads();

	if (cx < w && cy < h && cz < c) {
		data_c[addr] = data_a[addr] + share_b[threadIdx.z];
	}
}

__global__ void __sum_gradient_1d(
	const nn_type* a,
	nn_type* b,
	cuint n,
	cuint c
) {
	cuint cx = blockIdx.x * blockDim.x + threadIdx.x;
	cuint tidx = threadIdx.y * BLOCK_32 + threadIdx.x;

	__shared__ nn_type sm[BLOCK_32 * BLOCK_32];
	
	sm[tidx] = 0.f;
	__syncthreads();

	for (uint i = 0; i < n; i += BLOCK_32) {
		cuint cy = threadIdx.y + i;

		if (cx < c && cy < n) {
			sm[tidx] += a[cy * c + cx];
		}
	}

#pragma unroll
	for (uint i = BLOCK_32 / 2; i > 0; i /= 2) {
		cuint half_side = (threadIdx.y + i) * BLOCK_32 + threadIdx.x;

		__syncthreads();
		if (threadIdx.y < i) sm[tidx] += sm[half_side];
	}

	if (cx < c && threadIdx.y == 0) b[cx] = sm[threadIdx.x];
}

__global__ void __sum_gradient_2d(
	const nn_type* a,
	nn_type* b,
	cuint n,
	cuint h,
	cuint w,
	cuint c
) {
	/*
	threads = [4, 16, 16]
	blocks = [c]
	*/

	__shared__ nn_type sm[BLOCK_4 * BLOCK_16 * BLOCK_16];

	cuint tidx = threadIdx.z * (BLOCK_16 * BLOCK_16) + threadIdx.y * BLOCK_16 + threadIdx.x;
	cuint cidx = blockIdx.x;

	sm[tidx] = 0.f;
	__syncthreads();

	for (uint z = 0; z < n; z += BLOCK_4) {
		cuint tz = threadIdx.z + z;
		cuint nidx = tz * (c * h * w);

		for (uint y = 0; y < h; y += BLOCK_16) {
			cuint ty = threadIdx.y + y;
			cuint yidx = ty * (w * c);

			for (uint x = 0; x < w; x += BLOCK_16) {
				cuint tx = threadIdx.x + x;
				cuint xidx = tx * c;

				if (tz < n && ty < h && tx < w) sm[tidx] += a[nidx + yidx + xidx + cidx];
			}
		}
	}

#pragma unroll
	for (uint i = BLOCK_1024 / 2; i > 0; i /= 2) {
		__syncthreads();

		if (tidx < i) sm[tidx] += sm[tidx + i];
	}

	b[blockIdx.x] = sm[0];
}

__global__ void __add_tensor(
	const nn_type* a_input,
	const nn_type* b_input,
	nn_type* output,
	cuint size
) {
	cuint idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < size) output[idx] = a_input[idx] + b_input[idx];
}

__global__ void __add_tensor2(
	const nn_type* a_input,
	const nn_type b_input,
	nn_type* output,
	cuint size
) {
	cuint idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < size) output[idx] = a_input[idx] + b_input;
}

__global__ void __sub_tensor(
	const nn_type* a_input,
	const nn_type* b_input,
	nn_type* output,
	cuint size
) {
	cuint idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < size) output[idx] = a_input[idx] - b_input[idx];
}

__global__ void __sub_tensor2(
	const nn_type* a_input,
	const nn_type b_input,
	nn_type* output,
	cuint size
) {
	cuint idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < size) output[idx] = a_input[idx] - b_input;
}

__global__ void __sub_tensor3(
	const nn_type a_input,
	const nn_type* b_input,
	nn_type* output,
	cuint size
) {
	cuint idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < size) output[idx] = a_input - b_input[idx];
}

__global__ void __mul_tensor(
	const nn_type* a_input,
	const nn_type* b_input,
	nn_type* output,
	cuint size
) {
	cuint idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < size) output[idx] = a_input[idx] * b_input[idx];
}

__global__ void __mul_tensor2(
	const nn_type* a_input,
	const nn_type b_input,
	nn_type* output,
	cuint size
) {
	cuint idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < size) output[idx] = a_input[idx] * b_input;
}

__global__ void __div_tensor(
	const nn_type* a_input,
	const nn_type* b_input,
	nn_type* output,
	cuint size
) {
	cuint idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < size) output[idx] = a_input[idx] / b_input[idx];
}

__global__ void __div_tensor2(
	const nn_type* a_input,
	const nn_type b_input,
	nn_type* output,
	cuint size
) {
	cuint idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < size) output[idx] = a_input[idx] / b_input;
}

__global__ void __inv_tensor(
	const nn_type* a_input,
	nn_type* output,
	cuint size
) {
	cuint idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < size) output[idx] = 1. / a_input[idx];
}

/*******************************************

			    host functions

*******************************************/

void set_const_mem(cuint* h_mem, size_t len, size_t offset) {
#if _DEBUG
	check_cuda(hipMemcpyToSymbol(HIP_SYMBOL(__cmem), h_mem, sizeof(uint) * len, sizeof(uint) * offset));
#else
	hipMemcpyToSymbol(HIP_SYMBOL(__cmem), h_mem, sizeof(uint) * len, sizeof(uint) * offset);
#endif
}

cuint* get_const_mem(size_t len, size_t offset) {
	uint* ptr = NULL;

#if _DEBUG
	check_cuda(hipGetSymbolAddress((void**)&ptr, __cmem));
#else
	hipGetSymbolAddress((void**)&ptr, __cmem);
#endif

	return ptr + offset;
}

void transpose_param_init(
	const GpuTensor<nn_type>& input,
	const std::vector<uint>& ranks,
	cuint** c_dims,
	cuint** c_steps,
	cuint** c_ranks
) {
	const NN_Shape shape = input.get_shape();
	uint* ptr = new uint[shape.ranks()];
	size_t offset = 0;
	size_t len = (size_t)shape.ranks();

	int i = 0;

	for (const int& n : shape) ptr[i++] = (uint)n;

	set_const_mem(ptr, len, offset);
	*c_dims = get_const_mem(len, offset);

	offset += len;

	uint step = 1;

	while (i) {
		--i;

		cuint dim = ptr[i];
		ptr[i] = step;
		step *= dim;
	}

	set_const_mem(ptr, len, offset);
	*c_steps = get_const_mem(len, offset);

	offset += len;

	for (const int& n : ranks) ptr[i++] = n;

	set_const_mem(ptr, len, offset);
	*c_ranks = get_const_mem(len, offset);

	delete[] ptr;
}

void transpose(
	const GpuTensor<nn_type>& input,
	GpuTensor<nn_type>& output,
	const std::vector<uint>& ranks
) {
	dim3 threads(BLOCK_1024);
	dim3 blocks = get_grid_size(threads, (uint)input.get_shape().total_size());

	cuint* c_dims = NULL;
	cuint* c_steps = NULL;
	cuint* c_trans_ranks = NULL;

	transpose_param_init(input, ranks, &c_dims, &c_steps, &c_trans_ranks);

	__transpose<<<blocks, threads>>>(
		(nn_type*)input.get_ptr(),
		(nn_type*)output.get_ptr(),
		c_trans_ranks,
		c_dims,
		c_steps,
		(uint)input.get_shape().ranks(),
		(uint)input.get_shape().total_size()
	);
#if _DEBUG
	check_cuda(hipDeviceSynchronize());
	check_cuda(hipGetLastError());
#endif
}

void padding_dilation(
	hipStream_t s,
	const nn_type* input,
	nn_type* output,
	const NN_Tensor4dShape& in,
	const NN_Tensor4dShape& out,
	cuint offset_x,
	cuint offset_y,
	cuint stride_x,
	cuint stride_y
) {
	dim3 threads(BLOCK_32, BLOCK_32);
	dim3 blocks = get_grid_size(threads, in._w, in._h, in._c);

	__padding_dilation_2d<<<blocks, threads, 0, s>>>(
		input,
		output,
		(uint)in._w,
		(uint)in._h,
		(uint)in._c,
		(uint)out._w,
		(uint)out._h,
		stride_x,
		stride_y,
		offset_x,
		offset_y
	);
}

void add_bias_1d(
	const GpuTensor<nn_type>& input,
	const GpuTensor<nn_type>& bias,
	GpuTensor<nn_type>& output
) {
	const NN_Shape& in = input.get_shape();

	dim3 threads(BLOCK_32, BLOCK_32);
	dim3 blocks = get_grid_size(threads, in[1], in[0]);

	__add_bias_32x32<<<blocks, threads>>>(
		input.get_ptr(),
		bias.get_ptr(),
		output.get_ptr(),
		(uint)in[0],
		(uint)in[1]
	);
#if _DEBUG
	check_cuda(hipDeviceSynchronize());
	check_cuda(hipGetLastError());
#endif
}

void add_bias_2d(
	NN_Stream& s,
	const GpuTensor<nn_type>& input,
	const GpuTensor<nn_type>& bias,
	GpuTensor<nn_type>& output
) {
	const NN_Tensor4dShape& in = input.get_shape().get_4d_shape();
	const nn_type* in_data = input.get_ptr();
	const nn_type* bias_data = bias.get_ptr();
	nn_type* out_data = output.get_ptr();

	hipStream_t* p_st = s.get_stream();

	if (in._h >= BLOCK_16 && in._w >= BLOCK_16 || in._c <= BLOCK_4) {
		dim3 threads(BLOCK_16, BLOCK_16, BLOCK_4);
		dim3 blocks = get_grid_size(threads, in._w, in._h, in._c);

		for (int i = 0; i < in._n; ++i) {
			cuint index = in._h * in._w * in._c * i;
			const nn_type* d_in = in_data + index;
			nn_type* d_out = out_data + index;
			
			__add_bias_16x16x4<<<blocks, threads, 0, p_st[i % STREAMS]>>>(
				d_in,
				bias_data,
				d_out,
				(uint)in._h,
				(uint)in._w,
				(uint)in._c
			);
		}
	}
	else {
		dim3 threads(BLOCK_8, BLOCK_8, BLOCK_16);
		dim3 blocks = get_grid_size(threads, in._w, in._h, in._c);

		for (int i = 0; i < in._n; ++i) {
			cuint index = in._h * in._w * in._c;
			const nn_type* d_in = in_data + index;
			nn_type* d_out = out_data + index;

			__add_bias_8x8x16<<<blocks, threads, 0, p_st[i % STREAMS]>>>(
				d_in,
				bias_data,
				d_out,
				(uint)in._h,
				(uint)in._w,
				(uint)in._c
			);
		}
	}
}

void sum_gradient_1d(
	const GpuTensor<nn_type>& input,
	GpuTensor<nn_type>& output
) {
	const NN_Shape& in = input.get_shape();

	dim3 threads(BLOCK_32, BLOCK_32);
	dim3 blocks = get_grid_size(threads, in[1]);

	__sum_gradient_1d<<<blocks, threads>>>(
		input.get_ptr(),
		output.get_ptr(),
		(uint)in[0],
		(uint)in[1]
	);
}

void sum_gradient_2d(
	const GpuTensor<nn_type>& input,
	GpuTensor<nn_type>& output
) {
	const NN_Tensor4dShape& in = input.get_shape().get_4d_shape();

	dim3 threads(BLOCK_16, BLOCK_16, BLOCK_4);
	dim3 blocks(in._c);

	__sum_gradient_2d<<<blocks, threads>>>(
		input.get_ptr(),
		output.get_ptr(),
		(uint)in._n,
		(uint)in._h,
		(uint)in._w,
		(uint)in._c
	);
}

void add_tensor(
	const GpuTensor<nn_type>& a_input,
	const GpuTensor<nn_type>& b_input,
	GpuTensor<nn_type>& output
) {
	const NN_Shape& a_shape = a_input.get_shape();
	const NN_Shape& b_shape = b_input.get_shape();
	const NN_Shape& out_shape = output.get_shape();

	if (a_shape != b_shape || b_shape != out_shape) {
		ErrorExcept(
			"[add_tensor] shapes of input and output are different. %s != %s != %s",
			shape_to_str(a_shape),
			shape_to_str(b_shape),
			shape_to_str(out_shape)
		);
	}

	cuint len = (uint)a_shape.total_size();
	const dim3 threads(BLOCK_1024);
	const dim3 blocks = get_grid_size(threads, len);

	__add_tensor<<<blocks, threads>>>(
		a_input.get_ptr(),
		b_input.get_ptr(),
		output.get_ptr(),
		len
	);
}

void add_tensor(
	const GpuTensor<nn_type>& input,
	const nn_type scalar,
	GpuTensor<nn_type>& output
) {
	const NN_Shape& in_shape = input.get_shape();
	const NN_Shape& out_shape = output.get_shape();

	if (in_shape != out_shape) {
		ErrorExcept(
			"[add_tensor] shapes of input and output are different. %s != %s",
			shape_to_str(in_shape),
			shape_to_str(out_shape)
		);
	}

	cuint len = (uint)input.get_shape().total_size();
	const dim3 threads(BLOCK_1024);
	const dim3 blocks = get_grid_size(threads, len);

	__add_tensor2<<<blocks, threads>>>(
		input.get_ptr(),
		scalar,
		output.get_ptr(),
		len
	);
}

void sub_tensor(
	const GpuTensor<nn_type>& a_input,
	const GpuTensor<nn_type>& b_input,
	GpuTensor<nn_type>& output
) {
	const NN_Shape& a_shape = a_input.get_shape();
	const NN_Shape& b_shape = b_input.get_shape();
	const NN_Shape& out_shape = output.get_shape();

	if (a_shape != b_shape || a_shape != out_shape) {
		ErrorExcept(
			"[sub_tensor] shapes of input and output are different. %s != %s != %s",
			shape_to_str(a_shape),
			shape_to_str(b_shape),
			shape_to_str(out_shape)
		);
	}

	cuint len = (uint)a_shape.total_size();
	const dim3 threads(BLOCK_1024);
	const dim3 blocks = get_grid_size(threads, len);

	__sub_tensor<<<blocks, threads>>>(
		a_input.get_ptr(),
		b_input.get_ptr(),
		output.get_ptr(),
		len
	);
}

void sub_tensor(
	const GpuTensor<nn_type>& input,
	const nn_type scalar,
	GpuTensor<nn_type>& output
) {
	const NN_Shape& in_shape = input.get_shape();
	const NN_Shape& out_shape = output.get_shape();

	if (in_shape != out_shape) {
		ErrorExcept(
			"[sub_tensor] shapes of input and output are different. %s != %s",
			shape_to_str(in_shape),
			shape_to_str(out_shape)
		);
	}

	cuint len = (uint)in_shape.total_size();
	const dim3 threads(BLOCK_1024);
	const dim3 blocks = get_grid_size(threads, len);

	__sub_tensor2<<<blocks, threads>>>(
		input.get_ptr(),
		scalar,
		output.get_ptr(),
		len
	);
}

void sub_tensor(
	const nn_type scalar,
	const GpuTensor<nn_type>& input,
	GpuTensor<nn_type>& output
) {

}

void mul_tensor(
	const GpuTensor<nn_type>& a_input,
	const GpuTensor<nn_type>& b_input,
	GpuTensor<nn_type>& output
) {

}

void mul_tensor(
	const GpuTensor<nn_type>& input,
	const nn_type scalar,
	GpuTensor<nn_type>& output
) {
	const NN_Shape shape = output.get_shape();
	cuint len = (uint)shape.total_size();

	dim3 threads(BLOCK_1024);
	dim3 blocks = get_grid_size(threads, len);

	__mul_tensor2<<<blocks, threads>>>(
		input.get_ptr(),
		scalar,
		output.get_ptr(),
		len
	);
}

void div_tensor(
	const GpuTensor<nn_type>& a_input,
	const GpuTensor<nn_type>& b_input,
	GpuTensor<nn_type>& output
) {
	const NN_Shape shape = output.get_shape();
	cuint len = (uint)shape.total_size();

	dim3 threads(BLOCK_1024);
	dim3 blocks = get_grid_size(threads, len);

	__div_tensor<<<blocks, threads>>>(
		a_input.get_ptr(),
		b_input.get_ptr(),
		output.get_ptr(),
		len
	);
}

void div_tensor(
	const GpuTensor<nn_type>& input,
	const nn_type scalar,
	GpuTensor<nn_type>& output
) {
	cuint len = (uint)output.get_shape().total_size();
	dim3 threads(BLOCK_1024);
	dim3 blocks = get_grid_size(threads, len);

	__div_tensor2<<<blocks, threads>>>(
		input.get_ptr(),
		scalar,
		output.get_ptr(),
		len
	);
#if _DEBUG
	check_cuda(hipDeviceSynchronize());
	check_cuda(hipGetLastError());
#endif
}

void inv_tensor(
	const GpuTensor<nn_type>& input,
	GpuTensor<nn_type>& output
) {
	cuint len = (uint)output.get_shape().total_size();
	dim3 threads(BLOCK_1024);
	dim3 blocks = get_grid_size(threads, len);

	__inv_tensor << <blocks, threads >> > (
		input.get_ptr(),
		output.get_ptr(),
		len
		);
}