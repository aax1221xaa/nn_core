#include "hip/hip_runtime.h"
#include "../cuda_source/cast.cuh"

#ifndef __HIPCC__
#define __HIPCC__
#endif

//#include <hip/device_functions.h>
#include <>

#ifdef FIX_MODE

template <typename dT, typename sT>
__global__ void __cast(
	void* dst,
	void* src,
	cuint elem_size
) {
	cuint index = blockIdx.x * blockDim.x + threadIdx.x;

	if (index < elem_size) ((dT*)dst)[index] = (dT)((sT*)src)[index];
}

dtype get_type(bool* data) {
	return dtype::boolean;
}

dtype get_type(char* data) {
	return dtype::int8;
}

dtype get_type(unsigned char* data) {
	return dtype::uint8;
}

dtype get_type(int* data) {
	return dtype::int32;
}

dtype get_type(unsigned int* data) {
	return dtype::uint32;
}

dtype get_type(long* data) {
	return dtype::int64;
}

dtype get_type(unsigned long* data) {
	return dtype::uint64;
}

dtype get_type(float* data) {
	return dtype::float32;
}

dtype get_type(double* data) {
	return dtype::float64;
}

template <typename _T>
void cast_function(hipStream_t s, void* dst, dtype src_type, void* src, cuint len) {
	dim3 threads(SQR_BLOCK_SIZE);
	dim3 blocks = get_grid_size(threads, len);

	switch (src_type)
	{
	case dtype::boolean:
		__cast<_T, bool> << <blocks, threads, 0, s >> > (dst, src, len);
		break;
	case dtype::int8:
		__cast<_T, char> << <blocks, threads, 0, s >> > (dst, src, len);
		break;
	case dtype::uint8:
		__cast<_T, unsigned char> << <blocks, threads, 0, s >> > (dst, src, len);
		break;
	case dtype::int32:
		__cast<_T, int> << <blocks, threads, 0, s >> > (dst, src, len);
		break;
	case dtype::uint32:
		__cast<_T, unsigned int> << <blocks, threads, 0, s >> > (dst, src, len);
		break;
	case dtype::int64:
		__cast<_T, long> << <blocks, threads, 0, s >> > (dst, src, len);
		break;
	case dtype::uint64:
		__cast<_T, unsigned long> << <blocks, threads, 0, s >> > (dst, src, len);
		break;
	case dtype::float32:
		__cast<_T, float> << <blocks, threads, 0, s >> > (dst, src, len);
		break;
	case dtype::float64:
		__cast<_T, double> << <blocks, threads, 0, s >> > (dst, src, len);
		break;
	default:
		break;
	}

	//check_cuda(hipStreamSynchronize(s));
	//check_cuda(hipGetLastError());
}

void type_cast(hipStream_t s, dtype dst_type, void* dst, dtype src_type, void* src, cuint len) {
	switch (dst_type)
	{
	case dtype::boolean:
		cast_function<bool>(s, dst, src_type, src, len);

		break;
	case dtype::int8:
		cast_function<char>(s, dst, src_type, src, len);

		break;
	case dtype::uint8:
		cast_function<unsigned char>(s, dst, src_type, src, len);

		break;
	case dtype::int32:
		cast_function<int>(s, dst, src_type, src, len);

		break;
	case dtype::uint32:
		cast_function<unsigned int>(s, dst, src_type, src, len);

		break;
	case dtype::int64:
		cast_function<long>(s, dst, src_type, src, len);

		break;
	case dtype::uint64:
		cast_function<unsigned long>(s, dst, src_type, src, len);

		break;
	case dtype::float32:
		cast_function<float>(s, dst, src_type, src, len);

		break;
	case dtype::float64:
		cast_function<double>(s, dst, src_type, src, len);

		break;
	default:
		break;
	}
}

#endif

#ifndef FIX_MODE
template <typename dT, typename sT>
__global__ void __cast(
	void* dst,
	void* src,
	const uint elem_size
) {
	const uint index = blockIdx.x * blockDim.x + threadIdx.x;

	if (index < elem_size) ((dT*)dst)[index] = ((sT*)src)[index];
}

template <typename _T>
void cast_function(
	hipStream_t s,
	void* dst,
	const int src_type,
	void* src,
	const uint len) 
{
	dim3 block(SQR_BLOCK_SIZE);
	dim3 grid = get_grid_size(block, len);

	switch (src_type)
	{
	case CV_8SC1:
		__cast<_T, char> << <block, grid, 0, s >> > (dst, src, len);
		break;
	case CV_8UC1:
		__cast<_T, uchar> << <block, grid, 0, s >> > (dst, src, len);
		break;
	case CV_16SC1:
		__cast<_T, short> << <block, grid, 0, s >> > (dst, src, len);
		break;
	case CV_16UC1:
		__cast<_T, ushort> << <block, grid, 0, s >> > (dst, src, len);
		break;
	case CV_32SC1:
		__cast<_T, int> << <block, grid, 0, s >> > (dst, src, len);
		break;
	case CV_32FC1:
		__cast<_T, float> << <block, grid, 0, s >> > (dst, src, len);
		break;
	case CV_64FC1:
		__cast<_T, double> << <block, grid, 0, s >> > (dst, src, len);
		break;
	default:
		break;
	}

	check_cuda(hipStreamSynchronize(s));
}

void type_cast(hipStream_t s,
	const int dst_type,
	void* dst,
	const int src_type,
	void* src,
	const uint len) 
{
	switch (dst_type)
	{
	case CV_8SC1:
		cast_function<char>(s, dst, src_type, src, len);
		break;
	case CV_8UC1:
		cast_function<uchar>(s, dst, src_type, src, len);
		break;
	case CV_16SC1:
		cast_function<short>(s, dst, src_type, src, len);
		break;
	case CV_16UC1:
		cast_function<ushort>(s, dst, src_type, src, len);
		break;
	case CV_32SC1:
		cast_function<int>(s, dst, src_type, src, len);
		break;
	case CV_32FC1:
		cast_function<float>(s, dst, src_type, src, len);
		break;
	case CV_64FC1:
		cast_function<double>(s, dst, src_type, src, len);
		break;
	default:
		break;
	}
}
#endif