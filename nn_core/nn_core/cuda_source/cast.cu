#include "hip/hip_runtime.h"
#include "../cuda_source/cast.cuh"

#ifndef __HIPCC__
#define __HIPCC__
#endif

#include <hip/device_functions.h>
#include <>

#ifdef FIX_MODE

template <typename dT, typename sT>
__global__ void __cast(
	sT* src,
	dT* dst,
	size_t elem_size
) {
	size_t index = blockIdx.x * blockDim.x + threadIdx.x;

	if (index < elem_size) dst[index] = dT(src[index]);
}

dtype get_type(bool* data) {
	return dtype::boolean;
}

dtype get_type(char* data) {
	return dtype::int8;
}

dtype get_type(unsigned char* data) {
	return dtype::uint8;
}

dtype get_type(int* data) {
	return dtype::int32;
}

dtype get_type(unsigned int* data) {
	return dtype::uint32;
}

dtype get_type(long* data) {
	return dtype::int64;
}

dtype get_type(unsigned long* data) {
	return dtype::uint64;
}

dtype get_type(float* data) {
	return dtype::float32;
}

dtype get_type(double* data) {
	return dtype::float64;
}

template <typename _T>
void cast_function(dtype src_type, void* src, _T* dst, size_t len) {
	dim3 threads(BLOCK_1024);
	dim3 blocks((BLOCK_1024 + len - 1) / BLOCK_1024);

	switch (src_type)
	{
	case dtype::boolean:
		__cast<<<blocks, threads>>>((bool*)src, dst, len);
		break;
	case dtype::int8:
		__cast<<<blocks, threads>>>((char*)src, dst, len);
		break;
	case dtype::uint8:
		__cast<<<blocks, threads>>>((unsigned char*)src, dst, len);
		break;
	case dtype::int32:
		__cast<<<blocks, threads>>>((int*)src, dst, len);
		break;
	case dtype::uint32:
		__cast<<<blocks, threads>>>((unsigned int*)src, dst, len);
		break;
	case dtype::int64:
		__cast<<<blocks, threads>>>((long*)src, dst, len);
		break;
	case dtype::uint64:
		__cast<<<blocks, threads>>>((unsigned long*)src, dst, len);
		break;
	case dtype::float32:
		__cast<<<blocks, threads>>>((float*)src, dst, len);
		break;
	case dtype::float64:
		__cast<<<blocks, threads>>>((double*)src, dst, len);
		break;
	default:
		break;
	}

	//check_cuda(hipStreamSynchronize(s));
	//check_cuda(hipGetLastError());
}

void type_cast(dtype src_type, void* src, dtype dst_type, void* dst, size_t len) {
	switch (dst_type)
	{
	case dtype::boolean:
		cast_function(src_type, src, (bool*)dst, len);
		break;
	case dtype::int8:
		cast_function(src_type, src, (char*)dst, len);
		break;
	case dtype::uint8:
		cast_function(src_type, src, (unsigned char*)dst, len);
		break;
	case dtype::int32:
		cast_function(src_type, src, (int*)dst, len);
		break;
	case dtype::uint32:
		cast_function(src_type, src, (unsigned int*)dst, len);
		break;
	case dtype::int64:
		cast_function(src_type, src, (long*)dst, len);
		break;
	case dtype::uint64:
		cast_function(src_type, src, (unsigned long*)dst, len);
		break;
	case dtype::float32:
		cast_function(src_type, src, (float*)dst, len);
		break;
	case dtype::float64:
		cast_function(src_type, src, (double*)dst, len);
		break;
	default:
		break;
	}
}

#endif

#ifndef FIX_MODE
template <typename dT, typename sT>
__global__ void __cast(
	void* dst,
	void* src,
	cuint elem_size
) {
	cuint index = blockIdx.x * blockDim.x + threadIdx.x;

	if (index < elem_size) ((dT*)dst)[index] = (dT)((sT*)src)[index];
}

dtype get_type(bool* data) {
	return dtype::boolean;
}

dtype get_type(char* data) {
	return dtype::int8;
}

dtype get_type(unsigned char* data) {
	return dtype::uint8;
}

dtype get_type(int* data) {
	return dtype::int32;
}

dtype get_type(unsigned int* data) {
	return dtype::uint32;
}

dtype get_type(long* data) {
	return dtype::int64;
}

dtype get_type(unsigned long* data) {
	return dtype::uint64;
}

dtype get_type(float* data) {
	return dtype::float32;
}

dtype get_type(double* data) {
	return dtype::float64;
}

template <typename _T>
void cast_function(hipStream_t s, void* dst, dtype src_type, void* src, cuint len) {
	dim3 threads(SQR_BLOCK_SIZE);
	dim3 blocks = get_grid_size(threads, len);

	switch (src_type)
	{
	case dtype::boolean:
		__cast<_T, bool> << <blocks, threads, 0, s >> > (dst, src, len);
		break;
	case dtype::int8:
		__cast<_T, char> << <blocks, threads, 0, s >> > (dst, src, len);
		break;
	case dtype::uint8:
		__cast<_T, unsigned char> << <blocks, threads, 0, s >> > (dst, src, len);
		break;
	case dtype::int32:
		__cast<_T, int> << <blocks, threads, 0, s >> > (dst, src, len);
		break;
	case dtype::uint32:
		__cast<_T, unsigned int> << <blocks, threads, 0, s >> > (dst, src, len);
		break;
	case dtype::int64:
		__cast<_T, long> << <blocks, threads, 0, s >> > (dst, src, len);
		break;
	case dtype::uint64:
		__cast<_T, unsigned long> << <blocks, threads, 0, s >> > (dst, src, len);
		break;
	case dtype::float32:
		__cast<_T, float> << <blocks, threads, 0, s >> > (dst, src, len);
		break;
	case dtype::float64:
		__cast<_T, double> << <blocks, threads, 0, s >> > (dst, src, len);
		break;
	default:
		break;
	}

	//check_cuda(hipStreamSynchronize(s));
	//check_cuda(hipGetLastError());
}

void type_cast(hipStream_t s, dtype dst_type, void* dst, dtype src_type, void* src, cuint len) {
	switch (dst_type)
	{
	case dtype::boolean:
		cast_function<bool>(s, dst, src_type, src, len);

		break;
	case dtype::int8:
		cast_function<char>(s, dst, src_type, src, len);

		break;
	case dtype::uint8:
		cast_function<unsigned char>(s, dst, src_type, src, len);

		break;
	case dtype::int32:
		cast_function<int>(s, dst, src_type, src, len);

		break;
	case dtype::uint32:
		cast_function<unsigned int>(s, dst, src_type, src, len);

		break;
	case dtype::int64:
		cast_function<long>(s, dst, src_type, src, len);

		break;
	case dtype::uint64:
		cast_function<unsigned long>(s, dst, src_type, src, len);

		break;
	case dtype::float32:
		cast_function<float>(s, dst, src_type, src, len);

		break;
	case dtype::float64:
		cast_function<double>(s, dst, src_type, src, len);

		break;
	default:
		break;
	}
}
#endif