#include "hip/hip_runtime.h"
﻿#include "convolution.cuh"

#ifndef __HIPCC__
#define __HIPCC__
#endif

#include <hip/device_functions.h>
#include <>


/**********************************************/
/*											  */
/*				 kernel function			  */
/*										      */
/**********************************************/

__constant__ uint __indices[CONST_ELEM_SIZE];


__global__ void __conv_2d(
	float* input,
	float* kernel,
	float* output,
	cuint in_w,
	cuint k_n,
	cuint k_w,
	cuint k_h,
	cuint k_c,
	cuint out_w,
	cuint out_h,
	cuint st_w,
	cuint st_h
) {
	cuint cx = blockIdx.x * blockDim.x + threadIdx.x;
	cuint cy = blockIdx.y * blockDim.y + threadIdx.y;
	cuint sidx = threadIdx.y * BLOCK_SIZE + threadIdx.x;

	cuint x0 = (cx % out_w) * st_w;
	cuint y0 = (cx / out_w) * st_h;

	cuint n = k_w * k_h * k_c;
	cuint k = out_w * out_h;

	__shared__ float share_in[BLOCK_SIZE * BLOCK_SIZE];
	__shared__ float share_k[BLOCK_SIZE * BLOCK_SIZE];

	float* p_input = input + (y0 * in_w + x0);
	float* p_kernel = kernel + (cy * k_w * k_h * k_c);

	float sum = 0.f;

	for (uint i = 0; i < n; i += BLOCK_SIZE) {
		__syncthreads();

		share_k[sidx] = (i + threadIdx.x) < n && cy < k_n ? p_kernel[threadIdx.x + i] : 0.f;
		share_in[sidx] = cx < k && (threadIdx.y + i) < n ? p_input[__indices[threadIdx.y + i]] : 0.f;

		__syncthreads();

#pragma unroll
		for (uint e = 0; e < BLOCK_SIZE; ++e) {
			sum += share_in[e * BLOCK_SIZE + threadIdx.x] * share_k[threadIdx.y * BLOCK_SIZE + e];
		}
	}

	if (cx < k && cy < k_n) {
		output[cy * k + cx] = sum;
	}
}

__global__ void __correl_2d(
	float* d_output,
	float* d_kernel,
	float* d_input,
	cuint dout_w,
	cuint dk_n,
	cuint dk_w,
	cuint dk_h,
	cuint dk_c,
	cuint din_w,
	cuint din_h
) {
	cuint cx = blockIdx.x * blockDim.x + threadIdx.x;
	cuint cy = blockIdx.y * blockDim.y + threadIdx.y;
	cuint sidx = threadIdx.y * BLOCK_SIZE + threadIdx.x;

	cuint x0 = cx % din_w;
	cuint y0 = cx / din_w;

	cuint n = dk_w * dk_h * dk_c;
	cuint tn = dk_w * dk_h * dk_n;
	cuint k = din_w * din_h;

	__shared__ float share_in[BLOCK_SIZE * BLOCK_SIZE];
	__shared__ float share_k[BLOCK_SIZE * BLOCK_SIZE];

	float* p_dout = d_output + (y0 * dout_w + x0);
	float* p_kernel = d_kernel + (cy * dk_w * dk_h);

	float sum = 0.f;

	for (uint i = 0; i < tn; i += BLOCK_SIZE) {
		__syncthreads();

		cuint wh = (threadIdx.x + i) % (dk_w * dk_h);
		cuint t_c = (threadIdx.x + i) / (dk_w * dk_h);
		float* pk = p_kernel + (t_c * dk_w * dk_h * dk_c);

		share_k[sidx] = (i + threadIdx.x) < tn && cy < dk_c ? pk[wh] : 0.f;
		share_in[sidx] = cx < k && (threadIdx.y + i) < tn ? p_dout[__indices[threadIdx.y + i]] : 0.f;

		__syncthreads();

#pragma unroll
		for (uint e = 0; e < BLOCK_SIZE; ++e) {
			sum += share_in[e * BLOCK_SIZE + threadIdx.x] * share_k[threadIdx.y * BLOCK_SIZE + e];
		}
	}

	if (cx < k && cy < dk_c) {
		d_input[cy * k + cx] = sum;
	}
}

__global__ void __kernel_conv_2d_32x32_c_ind(
	float* input,
	float* d_output,
	float* gradient,
	cuint input_h,
	cuint input_w,
	cuint input_c,
	cuint d_output_h,
	cuint d_output_w,
	cuint d_output_c,
	cuint gradient_h,
	cuint gradient_w
) {
	cuint n = d_output_h * d_output_w;
	cuint k = gradient_h * gradient_w * input_c;

	cuint cx = blockIdx.x * blockDim.x + threadIdx.x;
	cuint cy = blockIdx.y * blockDim.y + threadIdx.y;

	cuint x0 = cx % gradient_w;
	cuint y0 = (cx / gradient_w) % gradient_h;
	cuint c0 = cx / (gradient_h * gradient_w);

	cuint sidx = threadIdx.y * BLOCK_SIZE + threadIdx.x;

	__shared__ float sm_in[BLOCK_SIZE * BLOCK_SIZE];
	__shared__ float sm_dout[BLOCK_SIZE * BLOCK_SIZE];

	float* p_dout = d_output + (cy * d_output_h * d_output_w);
	float* p_in = input + (c0 * (input_h * input_w) + y0 * input_w + x0);

	float sum = 0.f;

	for (int i = 0; i < n; i += BLOCK_SIZE) {
		__syncthreads();

		sm_dout[sidx] = (threadIdx.x + i) < n && cy < d_output_c ? p_dout[threadIdx.x + i] : 0.f;
		sm_in[sidx] = cx < k && (threadIdx.y + i) < n ? p_in[__indices[threadIdx.y + i]] : 0.f;

		__syncthreads();

#pragma unroll
		for (int e = 0; e < BLOCK_SIZE; ++e) {
			sum += sm_dout[threadIdx.y * BLOCK_SIZE + e] * sm_in[e * BLOCK_SIZE + threadIdx.x];
		}
	}

	if (cx < k && cy < d_output_c) {
		gradient[cy * k + cx] += sum;
	}
}

__global__ void __kernel_conv_2d_32x32_g_ind(
	float* input,
	float* d_output,
	float* gradient,
	uint* input_indices,
	cuint input_h,
	cuint input_w,
	cuint input_c,
	cuint d_output_h,
	cuint d_output_w,
	cuint d_output_c,
	cuint gradient_h,
	cuint gradient_w
) {
	cuint n = d_output_h * d_output_w;
	cuint k = gradient_h * gradient_w * input_c;

	cuint cx = blockIdx.x + blockDim.x + threadIdx.x;
	cuint cy = blockIdx.y + blockDim.y + threadIdx.y;

	cuint x0 = cx % gradient_w;
	cuint y0 = (cx / gradient_w) % gradient_h;
	cuint c0 = cx / (gradient_h * gradient_w);

	cuint sidx = threadIdx.y * BLOCK_SIZE + threadIdx.x;

	__shared__ float sm_in[BLOCK_SIZE * BLOCK_SIZE];
	__shared__ float sm_dout[BLOCK_SIZE * BLOCK_SIZE];

	float* p_dout = d_output + (cy * d_output_h * d_output_w);
	float* p_in = input + (c0 * (input_h * input_w) + y0 * input_w + x0);

	float sum = 0.f;

	for (int i = 0; i < n; i += BLOCK_SIZE) {
		__syncthreads();

		sm_dout[sidx] = (threadIdx.x + i) < n && cy < d_output_c ? p_dout[threadIdx.x + i] : 0.f;
		sm_in[sidx] = cx < k && (threadIdx.y + i) < n ? p_in[input_indices[threadIdx.y + i]] : 0.f;

		__syncthreads();

#pragma unroll
		for (int e = 0; e < BLOCK_SIZE; ++e) {
			sum += sm_dout[threadIdx.y * BLOCK_SIZE + e] * sm_in[e * BLOCK_SIZE + threadIdx.x];
		}
	}

	if (cx < k && cy < n) {
		gradient[cy * k + cx] += sum;
	}
}

__global__ void __transpose(
	float* input,
	float* output,
	cuint n,
	cuint c,
	cuint h,
	cuint w
) {
	cuint tidx = blockIdx.x * blockDim.x + threadIdx.x;
	cuint k_idx = tidx % (w * h);
	cuint k_count = tidx / (w * h);

	cuint row = k_count % c;
	cuint col = k_count / c;

	float* p_out = output + (row * (w * h * n) + col * (w * h));

	if (tidx < (n * h * w * c)) {
		p_out[k_idx] = input[tidx];
	}
}

__global__ void __dilation_2d(
	float* input,
	float* output,
	cuint iw,
	cuint ih,
	cuint ic,
	cuint ow,
	cuint oh,
	cuint scale,
	cint offset_x,
	cint offset_y
) {
	cuint in_cx = blockIdx.x * blockDim.x + threadIdx.x;
	cuint in_cy = blockIdx.y * blockDim.y + threadIdx.y;

	cuint out_cx = in_cx * scale + offset_x;
	cuint out_cy = in_cy * scale + offset_y;

	for (int c = 0; c < ic; ++c) {
		if (in_cx < iw && in_cy < ih) {
			output[c * (ow * oh) + out_cy * ow + out_cx] = input[c * (iw * ih) + in_cy * iw + in_cx];
		}
	}
}



/**********************************************/
/*											  */
/*				  host function 			  */
/*										      */
/**********************************************/

/*                convolution_2d              */

int get_output_size(
	int input_size,
	int kernel_size,
	int pad_size,
	int stride
) {
	return (input_size + (2 * pad_size) - kernel_size) / stride + 1;
}

void check_conv_2d(
	const NN_Tensor& d_input,
	const NN_Tensor& d_kernel,
	const NN_Tensor& d_output,
	int st_w,
	int st_h
) {
	int out_h = get_output_size(d_input.shape[-2], d_kernel.shape[-2], 0, st_h);
	int out_w = get_output_size(d_input.shape[-1], d_kernel.shape[-1], 0, st_w);

	if (d_output.shape[-2] != out_h || d_output.shape[-1] != out_w) {
		ErrorExcept(
			"[check_conv_2d] invalid output dimension %s",
			d_output.shape.get_str()
		);
	}
	else if (d_kernel.shape[1] != d_input.shape[1] || d_kernel.shape[0] != d_output.shape[1]) {
		ErrorExcept(
			"[check_conv_2d] invalid channels input: %s, kernel: %s, output: %s",
			d_input.shape.get_str(),
			d_kernel.shape.get_str(),
			d_output.shape.get_str()
		);
	}
}

void conv_2d(
	hipStream_t& stream,
	const NN_Tensor& d_input,
	const NN_Tensor& d_kernel,
	NN_Tensor& d_output,
	int st_w,
	int st_h
) {
	check_conv_2d(
		d_input,
		d_kernel,
		d_output,
		st_w,
		st_h
	);

	uint batch = d_input.shape[0];
	uint kc = d_kernel.shape[1];
	uint kh = d_kernel.shape[2];
	uint kw = d_kernel.shape[3];
	uint *indices = new uint[kc * kh * kw];

	for (int c = 0; c < kc; ++c) {
		for (int h = 0; h < kh; ++h) {
			uint* p_indices = indices + (c * kh * kw);
			for (int w = 0; w < kw; ++w) {
				p_indices[h * kw + w] = (c * d_input.shape[2] * d_input.shape[3]) + (h * d_input.shape[3]) + w;
			}
		}
	}
	check_cuda(hipMemcpyToSymbol(HIP_SYMBOL(__indices), indices, sizeof(uint) * kc * kh * kw));

	dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
	dim3 blocks = get_grid_size(threads, d_output.shape[3] * d_output.shape[2], d_output.shape[1]);

	for (int i = 0; i < batch; ++i) {
		float* d_in = d_input.data + (i * d_input.shape[3] * d_input.shape[2] * d_input.shape[1]);
		float* d_out = d_output.data + (i * d_output.shape[3] * d_output.shape[2] * d_output.shape[1]);

		__conv_2d<<<blocks, threads, 0, stream>>>(
			d_in,
			d_kernel.data,
			d_out,
			d_input.shape[3],
			d_kernel.shape[0],
			d_kernel.shape[3],
			d_kernel.shape[2],
			d_kernel.shape[1],
			d_output.shape[3],
			d_output.shape[2],
			st_w,
			st_h
			);
	}
	check_cuda(hipStreamSynchronize(stream));
	delete[] indices;
}


/*             correlation_2d            */

void check_correl_2d(
	const NN_Tensor& d_doutput,
	const NN_Tensor& d_kernel,
	const NN_Tensor& d_dinput
) {
	int d_in_w = d_doutput.shape[3] - d_kernel.shape[3] + 1;
	int d_in_h = d_doutput.shape[2] - d_kernel.shape[2] + 1;

	if (
		d_doutput.shape[1] != d_kernel.shape[0] ||
		d_dinput.shape[1] != d_kernel.shape[1] ||
		d_dinput.shape[3] != d_in_w ||
		d_dinput.shape[2] != d_in_h
		) {
		ErrorExcept(
			"[check_correl_2d] invalid (d_output, kernel, d_input) size. d_doutput: %s, d_tkernel: %s, d_dinput: %s",
			d_doutput.shape.get_str(), d_kernel.shape.get_str(), d_dinput.shape.get_str()
		);
	}
}

void correl_2d(
	hipStream_t& stream,
	const NN_Tensor& d_doutput,
	const NN_Tensor& d_kernel,
	NN_Tensor& d_dinput
) {
	check_correl_2d(
		d_doutput,
		d_kernel,
		d_dinput
	);

	uint batch = d_doutput.shape[0];
	uint kn = d_kernel.shape[0];
	uint kh = d_kernel.shape[2];
	uint kw = d_kernel.shape[3];
	uint* indices = new uint[kn * kh * kw];

	for (int n = 0; n < kn; ++n) {
		uint* p_indices = indices + (n * kh * kw);
		for (int h = 0; h < kh; ++h) {
			for (int w = 0; w < kw; ++w) {
				p_indices[h * kw + w] = (n * d_doutput.shape[2] * d_doutput.shape[3]) + (kh - h - 1) * d_doutput.shape[3] + (kw - w - 1);
			}
		}
	}
	check_cuda(hipMemcpyToSymbol(HIP_SYMBOL(__indices), indices, sizeof(uint) * kn * kh * kw));

	dim3 threads = dim3(BLOCK_SIZE, BLOCK_SIZE);
	dim3 blocks = get_grid_size(threads, d_dinput.shape[3] * d_dinput.shape[2], d_dinput.shape[1]);

	for (int i = 0; i < batch; ++i) {
		float* d_dout = d_doutput.data + (i * d_doutput.shape[1] * d_doutput.shape[2] * d_doutput.shape[3]);
		float* d_din = d_dinput.data + (i * d_dinput.shape[1] * d_dinput.shape[2] * d_dinput.shape[3]);

		__correl_2d<<<blocks, threads, 0, stream>>>(
			d_dout,
			d_kernel.data,
			d_din,
			d_doutput.shape[3],
			d_kernel.shape[0],
			d_kernel.shape[3],
			d_kernel.shape[2],
			d_kernel.shape[1],
			d_dinput.shape[3],
			d_dinput.shape[2]
		);
	}
	check_cuda(hipStreamSynchronize(stream));
	delete[] indices;
}

/*            transpose             */	

void check_transpose(
	const NN_Tensor& d_input,
	const NN_Tensor& d_output
) {
	const NN_Shape& in_shape = d_input.shape;
	const NN_Shape& out_shape = d_output.shape;

	if (in_shape.len < 2 || out_shape.len < 2) {
		ErrorExcept(
			"[check_transpose] input, output tensor channels are smaller than 2. input: %s, output: %s",
			in_shape.get_str(),
			out_shape.get_str()
		);
	}

	if (in_shape[0] != out_shape[1] || in_shape[1] != out_shape[0]) {
		ErrorExcept(
			"[check_transpose] input, output tensor 0, 1 channels are invalid. input: %s, output: %s",
			in_shape.get_str(),
			out_shape.get_str()
		);
	}
}

void transpose(
	hipStream_t& stream,
	const NN_Tensor& d_input,
	NN_Tensor& d_output
) {
	check_transpose(d_input, d_output);

	dim3 threads(SQR_BLOCK_SIZE);
	dim3 blocks = get_grid_size(threads, d_input.get_elem_size());

	__transpose<<<blocks, threads, 0, stream>>>(
		d_input.data,
		d_output.data,
		d_input.shape[0],
		d_input.shape[1],
		d_input.shape[2],
		d_input.shape[3]
	);
	check_cuda(hipStreamSynchronize(stream));
}

/*            dilation_2d           */

void check_dilation_2d(
	const NN_Tensor& input,
	const NN_Tensor& output,
	uint scale,
	int offset_x,
	int offset_y
) {
	int out_w = input.shape[3] * scale + offset_x;
	int out_h = input.shape[2] * scale + offset_y;

	if (out_w > output.shape[3] || out_h > output.shape[2]) {
		ErrorExcept(
			"[check_dilation_2d] output is too small. output: %s, expect output: [%d, %d, %d, %d]",
			output.shape.get_str(),
			output.shape[0],
			output.shape[1],
			out_h,
			out_w
		);
	}
}

void dilation_2d(
	hipStream_t& stream,
	const NN_Tensor& d_input,
	NN_Tensor& d_output,
	uint scale,
	int offset_x,
	int offset_y
) {
	check_dilation_2d(
		d_input,
		d_output,
		scale,
		offset_x,
		offset_y
	);

	check_cuda(hipMemset(d_output.data, 0, sizeof(float) * d_output.get_elem_size()));

	dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
	dim3 blocks = get_grid_size(threads, d_input.shape[3], d_input.shape[2]);

	for (int i = 0; i < d_input.shape[0]; ++i) {
		float* d_in = d_input.data + (i * d_input.shape[3] * d_input.shape[2] * d_input.shape[1]);
		float* d_out = d_output.data + (i * d_output.shape[3] * d_output.shape[2] * d_output.shape[1]);

		__dilation_2d<<<blocks, threads, 0, stream>>>(
			d_in,
			d_out,
			d_input.shape[3],
			d_input.shape[2],
			d_input.shape[1],
			d_output.shape[3],
			d_output.shape[2],
			scale,
			offset_x,
			offset_y
			);
	}

	check_cuda(hipStreamSynchronize(stream));
}


/*          kernel_convolution_2d          */

void check_kernel_conv_2d(
	const NN_Tensor& d_doutput,
	const NN_Tensor& d_input,
	NN_Tensor& d_gradient
) {
	const NN_Shape& dout_shape = d_doutput.shape;
	const NN_Shape& in_shape = d_input.shape;
	const NN_Shape& grad_shape = d_gradient.shape;

	int in_h = in_shape[2] - dout_shape[2] + 1;
	int in_w = in_shape[3] - dout_shape[3] + 1;

	if (grad_shape[2] != in_h ||
		grad_shape[3] != in_w ||
		grad_shape[0] != dout_shape[1] ||
		grad_shape[1] != in_shape[1] ||
		in_shape[0] != dout_shape[0]) {

		ErrorExcept(
			"[check_kernel_conv_2d] invalid tensor arguments size. d_input: %s, d_doutput: %s, gradient: %s",
			in_shape.get_str(),
			dout_shape.get_str(),
			grad_shape.get_str()
		);
	}
}

void kernel_conv_2d(
	hipStream_t& stream,
	const NN_Tensor& d_doutput,
	const NN_Tensor& d_input,
	NN_Tensor& d_gradient
) {
	check_kernel_conv_2d(
		d_doutput,
		d_input,
		d_gradient
	);

	uint* indices = new uint[d_doutput.shape[2] * d_doutput.shape[3]];

	for (int h = 0; h < d_doutput.shape[2]; ++h) {
		for (int w = 0; w < d_doutput.shape[3]; ++w) {
			indices[h * d_doutput.shape[3] + w] = h * d_input.shape[3] + w;
		}
	}

	check_cuda(hipMemset(d_gradient.data, 0, sizeof(float) * d_gradient.get_elem_size()));

	dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
	dim3 blocks = get_grid_size(
		threads,
		d_gradient.shape[2] * d_gradient.shape[3] * d_gradient.shape[1],
		d_gradient.shape[0]
	);

	if (d_doutput.shape[2] * d_doutput.shape[3] < CONST_ELEM_SIZE) {

		check_cuda(hipMemcpyToSymbol(HIP_SYMBOL(__indices), indices, sizeof(uint) * d_doutput.shape[2] * d_doutput.shape[3]));

		for (uint i = 0; i < d_doutput.shape[0]; ++i) {
			float* d_in = d_input.data + (i * d_input.shape[3] * d_input.shape[2] * d_input.shape[1]);
			float* d_dout = d_doutput.data + (i * d_doutput.shape[3] * d_doutput.shape[2] * d_doutput.shape[1]);

			__kernel_conv_2d_32x32_c_ind<<<blocks, threads, 0, stream>>>(
				d_in,
				d_dout,
				d_gradient.data,
				d_input.shape[2],
				d_input.shape[3],
				d_input.shape[1],
				d_doutput.shape[2],
				d_doutput.shape[3],
				d_doutput.shape[1],
				d_gradient.shape[2],
				d_gradient.shape[3]
				);
			check_cuda(hipStreamSynchronize(stream));
		}
	}
	else {
		uint* d_indices = NULL;

		check_cuda(hipMalloc(&d_indices, sizeof(uint) * d_doutput.shape[2] * d_doutput.shape[3]));
		check_cuda(hipMemcpy(d_indices, indices, sizeof(uint) * d_doutput.shape[2] * d_doutput.shape[3], hipMemcpyHostToDevice));

		for (uint i = 0; i < d_doutput.shape[0]; ++i) {
			float* d_in = d_input.data + (i * d_input.shape[3] * d_input.shape[2] * d_input.shape[1]);
			float* d_dout = d_doutput.data + (i * d_doutput.shape[3] * d_doutput.shape[2] * d_doutput.shape[1]);

			__kernel_conv_2d_32x32_g_ind<<<blocks, threads, 0, stream>>>(
				d_in,
				d_dout,
				d_gradient.data,
				d_indices,
				d_input.shape[2],
				d_input.shape[3],
				d_input.shape[1],
				d_doutput.shape[2],
				d_doutput.shape[3],
				d_doutput.shape[1],
				d_gradient.shape[2],
				d_gradient.shape[3]
				);
			check_cuda(hipStreamSynchronize(stream));
		}

		check_cuda(hipFree(d_indices));
	}
	delete[] indices;
}