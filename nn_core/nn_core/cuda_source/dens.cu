#include "hip/hip_runtime.h"
#include "dens.cuh"

#ifndef __HIPCC__
#define __HIPCC__
#endif

#include <hip/device_functions.h>
#include <>


/**********************************************/
/*											  */
/*				 kernel function			  */
/*										      */
/**********************************************/

__global__ void __matmul(
	float* a,
	float* b,
	float* c,
	const uint m,
	const uint n,
	const uint k
) {
	uint cx = blockIdx.x * blockDim.x + threadIdx.x;
	uint cy = blockIdx.y * blockDim.y + threadIdx.y;

	__shared__ float sm_a[BLOCK_SIZE * BLOCK_SIZE];
	__shared__ float sm_b[BLOCK_SIZE * BLOCK_SIZE];

	uint tidx = threadIdx.y * BLOCK_SIZE + threadIdx.x;
	float val = 0.f;

	for (int i = 0; i < n; i += BLOCK_SIZE) {
		__syncthreads();

		sm_a[tidx] = (threadIdx.x + i) < n && cy < m ? a[cy * n + (threadIdx.x + i)] : 0.f;
		sm_b[tidx] = cx < k && (threadIdx.y + i) < n ? b[(threadIdx.y + i) * k + cx] : 0.f;

		__syncthreads();

		for (int e = 0; e < BLOCK_SIZE; ++e) {
			val += sm_a[threadIdx.y * BLOCK_SIZE + e] * sm_b[e * BLOCK_SIZE + threadIdx.x];
		}
	}

	if (cx < k && cy < m) {
		c[cy * k + cx] = val;
	}
}



/**********************************************/
/*											  */
/*				  host function 			  */
/*										      */
/**********************************************/

//void check_dens(
//	const NN_Tensor4D input,
//	const NN_Tensor4D weight,
//	const NN_Tensor4D output
//) {
//	if (input.n != output.n || input.c != weight.n || output.c != weight.c) {
//		ErrorExcept(
//			"[matmul_check] invalid matrix size input: %s, weight: %s, output: %s",
//			dim_to_str(input),
//			dim_to_str(weight),
//			dim_to_str(output)
//		);
//	}
//}

void dens(
	const hipStream_t st,
	const NN_Tensor4D input,
	const NN_Tensor4D weight,
	NN_Tensor4D output
) {
	//check_dens(input, weight, output);

	dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
	dim3 blocks = get_grid_size(threads, output.c, output.n);

	__matmul << <blocks, threads, 0, st >> > (
		input.data,
		weight.data,
		output.data,
		input.n,
		input.c,
		output.c
	);

	check_cuda(hipStreamSynchronize(st));
}