#include "hip/hip_runtime.h"
#define CUDA_API_PER_THREAD_DEFAULT_STEAM 
#include "softmax.cuh"

#ifndef __HIPCC__
#define __HIPCC__
#endif

#include <hip/device_functions.h>
#include <>


__global__ void __softmax(
	const nn_type* input,
	nn_type* output,
	cint c
) {
	__shared__ nn_type smem[BLOCK_1024];

	smem[threadIdx.x] = 0.f;
	__syncthreads();

	for (int i = 0; i < c; i += BLOCK_1024) {
		cint cx = i + threadIdx.x;

		smem[threadIdx.x] += cx < c ? __expf(input[cx]) : 0.f;
		__syncthreads();
	}

#pragma unroll
	for (int i = (BLOCK_1024 / 2); i > 0; i /= 2) {
		if (threadIdx.x < i) {
			smem[threadIdx.x] += smem[threadIdx.x + i];
		}
		__syncthreads();
	}

	for (int i = 0; i < c; i += BLOCK_1024) {
		cint cx = i + threadIdx.x;

		if (cx < c) output[cx] = __expf(input[cx]) / smem[0];
	}
}

NN_Softmax::NN_Softmax(const char* name) :
	NN_Layer(name)
{
}

void NN_Softmax::get_output_shape(const std::vector<NN_Shape>& input_shape, std::vector<NN_Shape>& output_shape) {
	output_shape.push_back(input_shape[0]);
}

void NN_Softmax::build(const std::vector<NN_Shape>& input_shape) {

}

void NN_Softmax::run_forward(NN_Stream& st, const std::vector<GpuTensor<nn_type>>& input, std::vector<GpuTensor<nn_type>>& output) {
	const NC in = input[0].get_shape().get_nc();

	const nn_type* in_data = input[0].get_ptr();
	nn_type* out_data = output[0].get_ptr();


	dim3 threads(BLOCK_1024);
	dim3 blocks(1);

	for (int n = 0; n < in.n; ++n) {
		const nn_type* m_in_data = in_data + (n * in.c);
		nn_type* m_out_data = out_data + (n * in.c);

		__softmax<<<blocks, threads, 0, st[n % STREAMS]>>>(
			m_in_data,
			m_out_data,
			in.c
		);
	}
}