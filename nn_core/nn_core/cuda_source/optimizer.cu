#include "hip/hip_runtime.h"
#define CUDA_API_PER_THREAD_DEFAULT_STEAM 
#include "optimizer.cuh"

#ifndef __HIPCC__
#define __HIPCC__
#endif

#include <hip/device_functions.h>
#include <>



/**********************************************/
/*											  */
/*				 kernel function			  */
/*										      */
/**********************************************/

__global__ void __sgd(
	float* gradient,
	float* weight,
	float* w_momentum,
	cuint w_len,
	float learn_rate,
	float momentum_rate
) {
	cuint idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < w_len) {
		float m = momentum_rate * w_momentum[idx] + learn_rate * gradient[idx];
		weight[idx] -= m;
		w_momentum[idx] = m;
	}
}

__global__ void __rms_prop(
	float* gradient,
	float* weight,
	float* g,
	cuint w_len,
	float learn_rate,
	float decay_rate
) {
	cuint idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < w_len) {
		float grad = gradient[idx];
		float _g = decay_rate * g[idx] + (1 - decay_rate) * __powf(grad, 2.f);
		
		weight[idx] -= learn_rate / __powf(_g + EPSILON, 0.5f) * grad;
		g[idx] = _g;
	}
}

__global__ void __adam(
	float* gradient,
	float* weight,
	float* square_g,
	float* decay_g,
	cuint w_len,
	float learn_rate,
	float beta_1,
	float beta_2
) {
	cuint idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < w_len) {
		float grad = gradient[idx];
		float m = beta_1 * decay_g[idx] + (1 - beta_1) * grad;
		float v = beta_2 * square_g[idx] + (1 - beta_2) * __powf(grad, 2.f);
		
		float _m = m / (1 - beta_1);
		float _v = v / (1 - beta_2);

		weight[idx] -= learn_rate / (_v + EPSILON) * _m;
		decay_g[idx] = m;
		square_g[idx] = v;
	}
}


/**********************************************/
/*                                            */
/*                NN_Optimizer                */
/*                                            */
/**********************************************/

NN_Optimizer::NN_Optimizer() {

}

NN_Optimizer::~NN_Optimizer() {

}

NN_Optimizer* NN_Optimizer::create(const std::vector<GpuTensor<nn_type>>& weights) {
	ErrorExcept(
		"[NN_Optimizer::create] make this function."
	);

	return NULL;
}

void NN_Optimizer::run(NN_Stream& st, const std::vector<GpuTensor<nn_type>>& gradient) {
	ErrorExcept(
		"[NN_Optimizer::run] make this function."
	);
}


/**********************************************/
/*                                            */
/*					   SGD                    */
/*                                            */
/**********************************************/

SGD::SGD(const std::vector<GpuTensor<nn_type>> weights) :
	_weights(weights),
	_l_rate(0.f),
	_m_rate(0.f)
{
	for (const GpuTensor<nn_type>& m_weight : weights) {
		const NN_Shape shape = m_weight.get_shape();

		_moments.push_back(GpuTensor<nn_type>::zeros(shape));
	}
}

SGD::SGD(float l_rate, float m_rate) :
	_l_rate(l_rate),
	_m_rate(m_rate)
{
}

NN_Optimizer* SGD::create(const std::vector<GpuTensor<nn_type>>& weights) {
	SGD* optimizer = new SGD(weights);

	optimizer->_l_rate = _l_rate;
	optimizer->_m_rate = _m_rate;

	return optimizer;
}

void SGD::run(NN_Stream& st, const std::vector<GpuTensor<nn_type>>& gradient) {
	dim3 threads(BLOCK_1024);

	for (size_t i = 0; i < _weights.size(); ++i) {
		size_t w_len = _weights[i].get_shape().total_size();
		dim3 blocks = get_grid_size(threads, w_len);

		__sgd<<<blocks, threads>>>(
			gradient[i].get_ptr(),
			_weights[i].get_ptr(),
			_moments[i].get_ptr(),
			w_len,
			_l_rate,
			_m_rate
		);
	}
}


/**********************************************/
/*                                            */
/*					 RmsProp                  */
/*                                            */
/**********************************************/

RmsProp::RmsProp(const std::vector<GpuTensor<nn_type>> weights) :
	_weights(weights),
	_d_rate(0.f),
	_l_rate(0.f)
{
	for (const GpuTensor<nn_type>& m_weight : weights) {
		const NN_Shape shape = m_weight.get_shape();

		_square_g.push_back(GpuTensor<nn_type>::zeros(shape));
	}
}

RmsProp::RmsProp(float d_rate, float l_rate) :
	_d_rate(d_rate),
	_l_rate(l_rate)
{
}

NN_Optimizer* RmsProp::create(const std::vector<GpuTensor<nn_type>>& weights) {
	RmsProp* optimizer = new RmsProp(weights);

	optimizer->_d_rate = _d_rate;
	optimizer->_l_rate = _l_rate;

	return optimizer;
}

void RmsProp::run(NN_Stream& st, const std::vector<GpuTensor<nn_type>>& gradient) {
	dim3 blocks(BLOCK_1024);
	
	for (size_t i = 0; i < _weights.size(); ++i) {
		size_t len = _weights[i].get_shape().total_size();
		dim3 threads = get_grid_size(blocks, len);

		__rms_prop<<<blocks, threads>>>(
			gradient[i].get_ptr(),
			_weights[i].get_ptr(),
			_square_g[i].get_ptr(),
			len,
			_l_rate,
			_d_rate
		);
	}
}


/**********************************************/
/*                                            */
/*					   Adam                   */
/*                                            */
/**********************************************/

Adam::Adam(const std::vector<GpuTensor<nn_type>> weights) :
	_weights(weights),
	_l_rate(0.f),
	_beta1(0.f),
	_beta2(0.f)
{
	for (const GpuTensor<nn_type>& m_weight : weights) {
		const NN_Shape shape = m_weight.get_shape();

		_square_g.push_back(GpuTensor<nn_type>::zeros(shape));
		_decay_g.push_back(GpuTensor<nn_type>::zeros(shape));
	}
}

Adam::Adam(float l_rate, float beta1, float beta2) :
	_l_rate(l_rate),
	_beta1(beta1),
	_beta2(beta2)
{
}

NN_Optimizer* Adam::create(const std::vector<GpuTensor<nn_type>>& weights) {
	Adam* optimizer = new Adam(weights);

	optimizer->_l_rate = _l_rate;
	optimizer->_beta1 = _beta1;
	optimizer->_beta2 = _beta2;

	return optimizer;
}

void Adam::run(NN_Stream& st, const std::vector<GpuTensor<nn_type>>& gradient) {
	dim3 blocks(BLOCK_1024);

	for (size_t i = 0; i < _weights.size(); ++i) {
		size_t len = _weights[i].get_shape().total_size();
		dim3 threads = get_grid_size(blocks, len);

		__adam<<<blocks, threads>>>(
			gradient[i].get_ptr(),
			_weights[i].get_ptr(),
			_square_g[i].get_ptr(),
			_decay_g[i].get_ptr(),
			len,
			_l_rate,
			_beta1,
			_beta2
		);
	}
}